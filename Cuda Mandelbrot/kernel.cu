#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <ccomplex>
#include <hip/hip_runtime.h>
#include <>

struct Color
{
	char r, g, b, a;
};


__global__ void render (Color * canvas, double offsetx, double offsety, double zoom);
__device__ int mandelbrot (double startReal, double startImag, int maximum);
__device__ Color spectrum (double n);

int const threads = 64; 

int main ()
{
	sf::RenderWindow window (sf::VideoMode (1920, 1080), "Mandelbrot", sf::Style::Fullscreen);

	Color * video_canvas;
	hipMalloc (&video_canvas, 1920 * 1080 * 4);

	sf::Color * canvas = new sf::Color[1920 * 1080];

	sf::Texture texture;
	texture.create (1920, 1080);

	double offset_x = 0;
	double offset_y = 0;
	double zoom = 1;

	while (window.isOpen ())
	{
		sf::Event event;
		while (window.pollEvent (event))
		{
			switch (event.type)
			{
			default:
				break;

			case sf::Event::Closed:
				window.close ();
			}
		}
		
		if (sf::Keyboard::isKeyPressed (sf::Keyboard::Dash)) zoom /= 0.94;
		if (sf::Keyboard::isKeyPressed (sf::Keyboard::Equal)) zoom *= 0.94;
		if (sf::Keyboard::isKeyPressed (sf::Keyboard::A)) offset_x -= 0.01 * zoom;
		if (sf::Keyboard::isKeyPressed (sf::Keyboard::D)) offset_x += 0.01 * zoom;
		if (sf::Keyboard::isKeyPressed (sf::Keyboard::S)) offset_y += 0.01 * zoom;
		if (sf::Keyboard::isKeyPressed (sf::Keyboard::W)) offset_y -= 0.01 * zoom;


		if (sf::Keyboard::isKeyPressed (sf::Keyboard::Escape)) window.close();

		render <<< 1920 * 1080 / threads, threads >>> (video_canvas, offset_x, offset_y, zoom);

		hipMemcpy (canvas, video_canvas, 1920 * 1080 * 4, hipMemcpyDeviceToHost);
		
		texture.update (reinterpret_cast<sf::Uint8 *>(canvas), 1920, 1080, 0, 0);

		window.draw (sf::Sprite (texture));
		window.display ();
	}

	delete[] canvas;
	hipFree (video_canvas);
}

__global__ void render (Color * canvas, double offsetx, double offsety, double zoom)
{
	int const i = blockIdx.x * blockDim.x + threadIdx.x;
	int2 const p = { i % 1920, i / 1920 };

	int limit = 128;
	int limit_sqrt = sqrtf (limit);

	double f_x = (static_cast <double> (p.x) / 1080 * 2 - 1) * zoom + offsetx;
	double f_y = (static_cast <double> (p.y) / 1090 * 2 - 1) * zoom + offsety;

	int iters = mandelbrot (f_x, f_y, limit);

	Color color = spectrum (iters);

	canvas[1920 * p.y + p.x] = color;
}

__device__ int mandelbrot (double startReal, double startImag, int maximum) {
	int counter = 0;
	double zReal = startReal;
	double zImag = startImag;
	double nextRe;

	while (zReal * zReal + zImag * zImag <= 4.0 && counter <= maximum) {
		nextRe = zReal * zReal - zImag * zImag + startReal;
		zImag = 2.0 * zReal * zImag + startImag;
		zReal = nextRe;
		if (zReal == startReal && zImag == startImag) { // a repetition indicates that the point is in the Mandelbrot set
			return -1; // points in the Mandelbrot set are represented by a return value of -1
		}
		counter += 1;
	}

	if (counter >= maximum) {
		return -1; // -1 is used here to indicate that the point lies within the Mandelbrot set
	}
	else {
		return counter; // returning the number of iterations allows for colouring
	}
}

__device__ Color spectrum (double iterations)
{
	int r, g, b;

	if (iterations == -1) {
		r = 0;
		g = 0;
		b = 0;
	}
	else if (iterations == 0) {
		r = 255;
		g = 0;
		b = 0;
	}
	else {
		// colour gradient:      Red -> Blue -> Green -> Red -> Black
		// corresponding values:  0  ->  16  ->  32   -> 64  ->  127 (or -1)
		if (iterations < 16) {
			r = 16 * (16 - iterations);
			g = 0;
			b = 16 * iterations - 1;
		}
		else if (iterations < 32) {
			r = 0;
			g = 16 * (iterations - 16);
			b = 16 * (32 - iterations) - 1;
		}
		else if (iterations < 64) {
			r = 8 * (iterations - 32);
			g = 8 * (64 - iterations) - 1;
			b = 0;
		}
		else { // range is 64 - 127
			r = 255 - (iterations - 64) * 4;
			g = 0;
			b = 0;
		}
	}

	return { r, g, b, 255 };
}
